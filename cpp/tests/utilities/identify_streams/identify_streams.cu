/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <cxxabi.h>
#include <dlfcn.h>
#include <execinfo.h>
#include <iostream>

/*
  Print the stack trace from the current frame.
  Adapted from from https://panthema.net/2008/0901-stacktrace-demangled/
*/
__host__ void print_trace()
{
#ifdef __GNUC__
  // Try to get the stack trace.
  constexpr int kMaxStackDepth = 64;
  void* stack[kMaxStackDepth];
  auto depth   = backtrace(stack, kMaxStackDepth);
  auto strings = backtrace_symbols(stack, depth);

  if (strings == nullptr) {
    std::cout << "No stack trace could be found!" << std::endl;
  } else {
    // If we were able to extract a trace, parse it, demangle symbols, and
    // print a readable output.

    // allocate string which will be filled with the demangled function name
    size_t funcnamesize = 256;
    char* funcname      = (char*)malloc(funcnamesize);

    // Start at frame 1 to skip print_trace itself.
    for (int i = 1; i < depth; ++i) {
      char* begin_name   = nullptr;
      char* begin_offset = nullptr;
      char* end_offset   = nullptr;

      // find parentheses and +address offset surrounding the mangled name:
      // ./module(function+0x15c) [0x8048a6d]
      for (char* p = strings[i]; *p; ++p) {
        if (*p == '(') {
          begin_name = p;
        } else if (*p == '+') {
          begin_offset = p;
        } else if (*p == ')' && begin_offset) {
          end_offset = p;
          break;
        }
      }

      if (begin_name && begin_offset && end_offset && begin_name < begin_offset) {
        *begin_name++   = '\0';
        *begin_offset++ = '\0';
        *end_offset     = '\0';

        // mangled name is now in [begin_name, begin_offset) and caller offset
        // in [begin_offset, end_offset). now apply __cxa_demangle():

        int status;
        char* ret = abi::__cxa_demangle(begin_name, funcname, &funcnamesize, &status);
        if (status == 0) {
          funcname = ret;  // use possibly realloc()-ed string (__cxa_demangle may realloc funcname)
          std::cout << "#" << i << " in " << strings[i] << " : " << funcname << "+" << begin_offset
                    << std::endl;
        } else {
          // demangling failed. Output function name as a C function with no arguments.
          std::cout << "#" << i << " in " << strings[i] << " : " << begin_name << "()+"
                    << begin_offset << std::endl;
        }
      } else {
        std::cout << "#" << i << " in " << strings[i] << std::endl;
      }
    }

    free(funcname);
  }
  free(strings);
#else
  std::cout << "Backtraces are only support on GNU systems." << std::endl;
#endif  // __GNUC__
}

// clang-format off
/*
   We need to overload all the functions from the runtime API (assuming that we
   don't use the driver API) that accept streams. Here's a complete listing of
   the API pages that contain any APIs using streams as of 9/20/2022:
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__STREAM.html
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EVENT.html#group__CUDART__EVENT
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EXTRES__INTEROP.html#group__CUDART__EXTRES__INTEROP
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EXECUTION.html#group__CUDART__EXECUTION
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html#group__CUDART__MEMORY
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY__POOLS.html#group__CUDART__MEMORY__POOLS
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__OPENGL__DEPRECATED.html#group__CUDART__OPENGL__DEPRECATED
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EGL.html#group__CUDART__EGL
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__GRAPH.html#group__CUDART__GRAPH
   - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__HIGHLEVEL.html#group__CUDART__HIGHLEVEL
 */
// clang-format on

using cudaLaunchKernel_t = hipError_t (*)(const void*, dim3, dim3, void**, size_t, hipStream_t);

static cudaLaunchKernel_t cudaLaunchKernel_original;

void __attribute__((constructor)) init();
void init()
{
  cudaLaunchKernel_original = (cudaLaunchKernel_t)dlsym(RTLD_NEXT, "hipLaunchKernel");
}

__host__ hipError_t hipLaunchKernel(
  const void* func, dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream)
{
  if (stream == static_cast<hipStream_t>(0) || (stream == hipStreamLegacy) ||
      (stream == hipStreamPerThread)) {
    std::cout << "Found unexpected default stream!" << std::endl;
    print_trace();
    std::cout << std::endl;
  }
  return cudaLaunchKernel_original(func, gridDim, blockDim, args, sharedMem, stream);
}
